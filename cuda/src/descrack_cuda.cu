#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iomanip>
#include <iostream>

#include "des.h"
#include "descrack_cuda_messages.hpp"
#include "input_descrack.h"


// Valori costanti per ogni kernel
static __constant__ DESBlock cipherTextBlockDevice = {0, 0};
static __constant__ DESBlock plainTextBlockDevice = {0, 0};
static __constant__ char alphabetDevice[128] = {0};
static __constant__ int alphabetLengthDevice = 1;
// Valori "restituiti" dal kernel
static __device__ bool keyFoundDevice = false;
static __device__ DESBlock keyDevice = {0, 0};


/**
 * Converte un valore ad una chiave specifica. 
 * Su CUDA, il valore utilizza un intero per il primo byte e 
 * gli identificatori per thread/blocco/griglia per gli altri.
 * 
 * @param keyLength lunghezza della chiave
 * @param value valore da convertire nel primo byte della chiave
 * 
 * @return la chiave relativa al valore in input e agli identificatori
 **/
__device__ static DESBlock valueToKey(const int keyLength, const int value) {
    DESBlock key = {0, 0};

    // È possibile utilizzare uno switch-case con un solo break alla fine
    // perchè ogni parte in basso è richiesta da quelle in alto
    switch (keyLength) {
        case 8:
            key.lo |= alphabetDevice[blockIdx.z / alphabetLengthDevice] << 1;
        case 7:
            key.lo |= alphabetDevice[blockIdx.y / alphabetLengthDevice] << 9;
        case 6:
            key.lo |= alphabetDevice[blockIdx.x / alphabetLengthDevice] << 17;
        case 5:
            key.lo |= alphabetDevice[blockIdx.z % alphabetLengthDevice] << 25;
        case 4:
            key.hi |= alphabetDevice[blockIdx.y % alphabetLengthDevice] << 1;
        case 3:
            key.hi |= alphabetDevice[blockIdx.x % alphabetLengthDevice] << 9;
        case 2:
            key.hi |= alphabetDevice[threadIdx.x] << 17;
        default:
            key.hi |= alphabetDevice[value] << 25;
    }
    return key;
}

/**
 * Verifica che la chiave sia quella da utilizzare per decifrare il blocco. 
 * Su CUDA, tutti gli altri valori necessari per la verifica risiedono nella memoria costante del device.
 * 
 * @param key riferimento alla chiave da testare
 * 
 * @return TRUE se la chiave è quella utilizzata per cifrare, FALSE altrimenti
 **/
__device__ static bool keyTest(const DESBlock &key) {
    DESBlock plainTextBlockTemp = decrypt(&cipherTextBlockDevice, &key);

    return plainTextBlockDevice.hi == plainTextBlockTemp.hi &&
           plainTextBlockDevice.lo == plainTextBlockTemp.lo;
}

/**
 * Kernel CUDA.
 * 
 * @param keyLength lunghezza della chiave da utilizzare per generare le chiavi
 **/
__global__ static void crackKey(const int keyLength) {
    DESBlock keyTemp = {0, 0};

    for (int i = 0; i < alphabetLengthDevice && !keyFoundDevice; i++) {
        keyTemp = valueToKey(keyLength, i);
        if (keyTest(keyTemp)) {
            keyFoundDevice = true;
            keyDevice = keyTemp;
        }
    }
}

int main(int argc, char **argv) {
    InputDESCrack input = inputDESCrackInit(argc, argv);
    DESBlock key = {0, 0};
    bool keyFound = false;
    dim3 gridSize = 1;
    int blockSize = 1;
    hipError_t error;

    // Controllo errori sul device, prima di iniziare
    error = hipGetLastError();
    if (error != hipSuccess)
        exitDuringDeviceInit(hipGetErrorString(error));
    // Copia i valori nella memoria costante
    // In caso di errore, stampa un messaggio ed esci
    // NOTA: CUDA permette di assegnare/leggere valori nella memoria del device tramite il loro "simbolo",
    //  cioè il nome la variabile, che deve risiedere nella stessa unità di compilazione
    error = hipMemcpyToSymbol(HIP_SYMBOL(cipherTextBlockDevice), &input.cipherTextBlock, sizeof(DESBlock));
    if (error != hipSuccess)
        exitDuringCopytoDevice("cipher text block", hipGetErrorString(error));
    error = hipMemcpyToSymbol(HIP_SYMBOL(plainTextBlockDevice), &input.plainTextBlock, sizeof(DESBlock));
    if (error != hipSuccess)
        exitDuringCopytoDevice("plain text block", hipGetErrorString(error));
    error = hipMemcpyToSymbol(HIP_SYMBOL(alphabetLengthDevice), &input.alphabetLength, sizeof(int));
    if (error != hipSuccess)
        exitDuringCopytoDevice("alphabet length value", hipGetErrorString(error));
    error = hipMemcpyToSymbol(HIP_SYMBOL(alphabetDevice), &input.alphabet, input.alphabetLength * sizeof(char));
    if (error != hipSuccess)
        exitDuringCopytoDevice("alphabet array", hipGetErrorString(error));
    // Se la chiave non è stata ancora trovata, ripeti per ogni valore della lunghezza della chiave...
    for (int i = input.minKeyLength; i <= input.maxKeyLength && !keyFound; i++) {
        // Imposta le dimensioni di blocco e griglia
        switch (i) {
            case 8:
                gridSize.z = input.alphabetLength * input.alphabetLength;
            case 7:
                gridSize.y = input.alphabetLength * input.alphabetLength;
            case 6:
                gridSize.x = input.alphabetLength * input.alphabetLength;
            case 5:
                gridSize.z = gridSize.z == 1 ? input.alphabetLength : gridSize.z;
            case 4:
                gridSize.y = gridSize.y == 1 ? input.alphabetLength : gridSize.y;
            case 3:
                gridSize.x = gridSize.x == 1 ? input.alphabetLength : gridSize.x;
            case 2:
                blockSize = input.alphabetLength;
            default:
                break;
        }
        // Lancia il kernel
        crackKey<<<gridSize, blockSize>>>(i);
        // Controllo errori durante il lancio del kernel
        error = hipGetLastError();
        if (error != hipSuccess)
            exitDuringKernelLaunch(hipGetErrorString(error));
        // Attendo il termine dell'esecuzione e controllo eventuali errori
        error = hipDeviceSynchronize();
        if (error != hipSuccess)
            exitDuringDeviceSynch(hipGetErrorString(error));
        // Copia il valore del flag dal device
        // In caso di errore, stampa un messaggio ed esci
        error = hipMemcpyFromSymbol(&keyFound, HIP_SYMBOL(keyFoundDevice), sizeof(bool));
        if (error != hipSuccess)
            exitDuringCopyFromDevice("\"found\" flag", hipGetErrorString(error));
        // Se la chiave è stata trovata...
        if (keyFound) {
            // ...copia il valore della chiave dal device
            // In caso di errore, stampa un messaggio ed esci
            error = hipMemcpyFromSymbol(&key, HIP_SYMBOL(keyDevice), sizeof(DESBlock));
            if (error != hipSuccess)
                exitDuringCopyFromDevice("key found", hipGetErrorString(error));
            // ...forza la parità (dispari) e stampala
            forceOddParity(&key);
            std::cout << std::setw(8) << std::setfill('0') << std::hex << key.hi
                      << std::setw(8) << std::setfill('0') << std::hex << key.lo
                      << std::endl;
        }
    }
    return EXIT_SUCCESS;
}